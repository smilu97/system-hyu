#include "hip/hip_runtime.h"

#include "matrix.h"

matrix * read_mat(const char * filepath)
{
     FILE * fd = fopen(filepath, "r");
     if(fd == NULL) {
         fprintf(stderr, "Failed to open matrix file\n");
         return NULL;
     }

     int cap = 16000000;
     int size = 0;
     float * arr = (float*)malloc(sizeof(float)*cap);
     if(arr == NULL) {
         fprintf(stderr, "Failed to allocate memory to load matrix\n");
         fclose(fd);
         return NULL;
     }

     int col_size = -1;
     int tmp;
     while(~fscanf(fd, "%d", &tmp)) {
         if(size >= cap) {
             float * narr = (float*)malloc(sizeof(float)*cap*2);
             if(narr == NULL) {
                 fprintf(stderr, "Matrix is too big!\n");
                 fclose(fd);
                 free(arr);
                 return NULL;
             }
             memcpy(narr, arr, sizeof(float)*size);
             free(arr);
             arr = narr;
         }
         arr[size++] = (float)tmp;
         if(fgetc(fd) == '\n' && col_size == -1) {
             col_size = size;
         }
     }
     fclose(fd);

     matrix * ret = create_mat(size / col_size, col_size);
     if(ret == NULL) {
         fprintf(stderr, "Failed to create_mat: read_mat\n");
         free(arr);
         return NULL;
     }
     memcpy(ret->v, arr, size);
     free(arr);

     return ret;
}

matrix * create_mat(int m, int n)
{
    matrix * ret = (matrix*)malloc(sizeof(matrix));
    if(ret == NULL) {
        fprintf(stderr, "Failed to allocate ret: create_mat\n");
        return NULL;
    }
    ret->v = (float*)malloc(sizeof(float)*m*n);
    if(ret->v == NULL) {
        fprintf(stderr, "Toob big too create: create_mat()");
        free(ret);
        return NULL;
    }
    ret->m = m;
    ret->n = n;
    
    return ret;
}

long long cuda_matmul(float * A, float * B, float * C, int m, int n, int l)
{
    float *cA, *cB, *cC;

    hipMalloc(&cA, sizeof(float)*m*n);
    hipMalloc(&cB, sizeof(float)*n*l);
    hipMalloc(&cC, sizeof(float)*m*l);

    hipMemcpy(cA, A, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(cB, B, sizeof(float)*n*l, hipMemcpyHostToDevice);

    dim3 dimBlock(m,l);
    dim3 dimGrid(1,1);

    cuda_matmul_unit<<<dimGrid,dimBlock>>>(cA,cB,cC,n,l);

    hipMemcpy(C, cC, sizeof(float)*m*l, hipMemcpyDeviceToHost);

    hipFree(cA);
    hipFree(cB);
    hipFree(cC);

    long long ret = 0;
    for(int i=0; i<m*l; ++i) ret += C[i];

    return ret;
}

__global__ void cuda_matmul_unit(float * A, float * B, float * C, int n, int l)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float ret = 0;

    for(int i=0; i<n; ++i) {
        ret += A[ty*n+i] * B[i*l+tx];
    }

    C[ty*l+tx] = ret;
}

long long mat_sum_element(matrix * mat)
{
    long long ret = 0;
    int size = mat->m * mat->n;
    for(int i=0; i<size; ++i) {
        ret += mat->v[i];
    }
    return ret;
}

